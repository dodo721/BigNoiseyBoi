#include "hip/hip_runtime.h"
#include "defines.hpp"

#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include "imgutils.hpp"
#include "colour_indexer.cuh"

using namespace std;

bool ColourEquals(colour* a, colour* b) {
    bool bEq = a[0] == b[0];
    bool gEq = a[1] == b[1];
    bool rEq = a[2] == b[2];
    return bEq && gEq && rEq;
}

__device__
bool ColourEqualsGPU(colour* a, colour* b) {
    bool bEq = a[0] == b[0];
    bool gEq = a[1] == b[1];
    bool rEq = a[2] == b[2];
    return bEq && gEq && rEq;
}

void IndexColours (colour** colours, int n, ColourEntry* indexer) {
    int indexCount = 0;
    for (int i = 0; i < n; i ++) {
        int idx = colours[i][0] + (colours[i][1] * 256) + (colours[i][2] * 65536);
        //printf("i: %d - Colour: %d,%d,%d; Index: %d\n", i, colours[colIdx], colours[colIdx + 1], colours[colIdx + 2], idx);
        indexer[idx].occupied = full_index;
        indexer[idx].order = indexCount;
        indexCount++;
        //atomicExch((int*)&indexer[idx], full_index);
    }
}

__device__
bool IndexIntersects(int index, colour3* indexer) {
    // TODO: Needed?
}

ColourEntry* initColourIndexer(ColourEntry* indexer_cpu) {
    ColourEntry* indexer;
    int indexer_size = sizeof(ColourEntry) * indexer_capacity;
    hipMallocManaged(&indexer, indexer_size);
    hipMemcpy(indexer, indexer_cpu, indexer_size, hipMemcpyHostToDevice);
    return indexer;
}

// Reverse pixel generation algorithm to figure out to what pixel each excluded colour belongs
ColourEntry* prepareExclusionList(colour* exclusions, int size) {

    double t = (double)cv::getTickCount();

    // Create shared memory indexer for indexing operations
    ColourEntry* indexer_cpu = new ColourEntry[indexer_capacity];

    colour** sortedExcl = new colour* [size];
    for (int i = 0; i < size; i++) {
        sortedExcl[i] = new colour[3];
        sortedExcl[i][0] = exclusions[i * 3];
        sortedExcl[i][1] = exclusions[i * 3 + 1];
        sortedExcl[i][2] = exclusions[i * 3 + 2];
    }
    sort(sortedExcl, sortedExcl + size, SortByColour);

    IndexColours (sortedExcl, size, indexer_cpu);

    // Use linear op to find order using CPU
    for (int i = 0; i < indexer_capacity; i++) {
        if (indexer_cpu[i].occupied) {
            //indexer_shared_mem[i].order = count;
            cout << "Order for index " << i << ": " << indexer_cpu[i].order << endl;
            //count++;
        }
    }

    // Create GPU-only indexer for use in image gen
    ColourEntry* indexer = initColourIndexer(indexer_cpu);

    t = ((double)cv::getTickCount() - t) / cv::getTickFrequency();
    cout << "Indexed in " << t << "s" << endl;

    delete[] indexer_cpu;

    return indexer;

}
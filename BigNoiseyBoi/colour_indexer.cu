#include "hip/hip_runtime.h"
#include "defines.hpp"

#include <iostream>
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include "imgutils.hpp"
#include "colour_indexer.cuh"

using namespace std;

bool ColourEquals(colour* a, colour* b) {
    bool bEq = a[0] == b[0];
    bool gEq = a[1] == b[1];
    bool rEq = a[2] == b[2];
    return bEq && gEq && rEq;
}

__device__
bool ColourEqualsGPU(colour* a, colour* b) {
    bool bEq = a[0] == b[0];
    bool gEq = a[1] == b[1];
    bool rEq = a[2] == b[2];
    return bEq && gEq && rEq;
}

void IndexColours (colour* colours, int n, ColourEntry* indexer) {
    int indexCount = 0;
    for (int i = 0; i < n; i ++) {
        int colIdx = i * 3;
        int idx = colours[colIdx] + (colours[colIdx + 1] * 256) + (colours[colIdx + 2] * 65536);
        //printf("i: %d - Colour: %d,%d,%d; Index: %d\n", i, colours[colIdx], colours[colIdx + 1], colours[colIdx + 2], idx);
        indexer[idx].occupied = full_index;
        indexer[idx].order = indexCount;
        indexCount++;
        //atomicExch((int*)&indexer[idx], full_index);
    }
}

ColourEntry* initColourIndexer(ColourEntry* indexer_cpu) {
    ColourEntry* indexer;
    int indexer_size = sizeof(ColourEntry) * indexer_capacity;
    hipMalloc(&indexer, indexer_size);
    hipMemcpy(indexer, indexer_cpu, indexer_size, hipMemcpyHostToDevice);
    return indexer;
}

// Reverse pixel generation algorithm to figure out to what pixel each excluded colour belongs
ColourEntry* prepareExclusionList(colour* exclusions, int size, bool gpu) {

    // Create shared memory indexer for indexing operations
    ColourEntry* indexer_cpu = new ColourEntry[indexer_capacity];

    IndexColours (exclusions, size, indexer_cpu);

    if (!gpu) return indexer_cpu;

    // Create GPU-only indexer for use in image gen
    ColourEntry* indexer = initColourIndexer(indexer_cpu);

    delete[] indexer_cpu;

    return indexer;

}

ColourEntry* CreateIndex(colour* exclusions, size_t exclLength, bool gpu, bool verbose) {
    if (verbose) cout << "Indexing " << exclLength << " colours" << endl;
    ColourEntry* exclusionIndex = nullptr;
    if (exclusions != nullptr) {
        double t = cv::getTickCount();
        exclusionIndex = prepareExclusionList(exclusions, exclLength, gpu);
        t = (cv::getTickCount() - t) / cv::getTickFrequency();
        if (verbose) cout << "Exclusions indexed in " << t << endl;
    }
    return exclusionIndex;
}

void FreeIndex(ColourEntry* index, bool gpu) {
    if (gpu)
        hipFree(index);
    else
        delete[] index;
}